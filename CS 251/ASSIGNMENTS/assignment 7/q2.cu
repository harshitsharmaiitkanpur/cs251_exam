
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 100000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array{
                    long long number;

};

__global__ void calculate(char *mem, long long num, long long i)
{
      long long z = (blockDim.x * blockIdx.x + threadIdx.x);
      long long size = z*i;

      if(size >= num)
        return;

      struct num_array *b = (struct num_array *)(mem + ((size+i-1) * sizeof(long long)));
      struct num_array *a = (struct num_array *)(mem + (size * sizeof(long long)));

        long long Xor;
        if(b->number!=-1)
		Xor = a->number ^ b->number;
        else
                Xor = a->number;

        if(z%2!=0)
		b->number = Xor;   
        else
                a->number = Xor;
      return;
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    long long c;
    struct num_array *pa;
    char *ptr;
    char *sptr;
    char *gpu_mem;
    long long num1 = NUM;
    long long blocks;
    long long seed;

    if(argc == 3)
    {
		num1 = atoi(argv[1]);
    	        seed = atoi(argv[2]);

    	        if(num1 <= 0)
        	   num1 = NUM;
    }

    blocks = num1 /1024;

    if(num1 % 1024)
        ++blocks;
    

    long long num2;
    num2 = num1;
    num1 = blocks*1024;

    srand(seed);
    ptr = (char *)malloc(num1 * sizeof(long long));
    sptr = ptr;

    for(c=0; c < num1; c++)
   {
       pa = (struct num_array *) sptr;
       pa->number = random();
       if(c>=num2)
                pa->number=-1;
       sptr += sizeof(long long);
    }


    gettimeofday(&t_start, NULL);

    hipMalloc(&gpu_mem, num1 * sizeof(long long));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num1 * sizeof(long long) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");

    gettimeofday(&start, NULL);

    long long block1;
    num1 = num2;

    for(long long j=2;j<2*num1;j=2*j)
{
        block1=blocks/j;
        if(blocks%j)
            block1++;

    calculate<<<block1, 1024>>>(gpu_mem, num1, j);
    CUDA_ERROR_EXIT("kernel invocation");

}

    gettimeofday(&end, NULL);

    hipMemcpy(ptr, gpu_mem, num1 * sizeof(long long) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);

    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;

    pa = (struct num_array *) (sptr);
    printf("result = %lld\n ", pa->number);

    free(ptr);
}


